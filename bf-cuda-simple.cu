#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "functions.h"
}

#define BLOCK_SIZE 32

/*
    Bellman-Ford algorithm, parallelized with CUDA, through a tiled kernel. 

    @param adj: adjacency matrix
    @param dist: distance array
    @param n: number of vertices
    @param negative: flag to indicate negative cycle
    @param tile_size: size of the tile, it must be V / N_THREADS + 1.
*/
__global__ void bellman_ford_tiled(int *adj, int *dist, int n, int *negative, int tile_size) {
    int v = threadIdx.x; 
    int loc_idx = v * tile_size;

    // First k-1 iterations
    for(int k=0; k<n-1; k++) {
        for(int u=0; u<n; u++) { // loop over the starting nodes
            for(int t=0; t<tile_size; t++) { // Loop over the arrival nodes in the tile
                
                if(loc_idx + t < n) {
                    if(dist[u] + adj[u * n + loc_idx + t] < dist[loc_idx + t]) {
                        dist[loc_idx + t] = dist[u] + adj[u * n + loc_idx + t];
                    }
                }

            }
        }
        __syncthreads();
    }

    // Check for negative cycles
    for(int u=0; u<n; u++) {
        for(int t=0; t<tile_size; t++) {
            if(v + t < n) {
                if(dist[u] + adj[u * n + loc_idx + t] < dist[loc_idx + t]) {
                    *negative = 1;
                }
            }
        }
    }
    
}

int main(int argc, char **argv) {

    char *size = argv[2];
    char *n_graphs = argv[1];
    char *threads = argv[3];

    char *gfile = (char *) malloc(100 * sizeof(char));
    char *ofile = (char *) malloc(100 * sizeof(char));
    char *zeros = (char *) malloc(100 * sizeof(char));

    int num_threads = atoi(threads);

    int tile_size = 1; 

    clock_t start, end;
    double avg_time = 0;

    if (num_threads > atoi(size)) num_threads=atoi(size);

    printf("Starting the loop\n");
    printf("\n");
    
    for (int i = 0; i < atoi(n_graphs); i++) {
        get_str(i, atoi(n_graphs), zeros);
        
        snprintf(gfile, 100, "./tests/graphs/%s_%s%d.txt", size, zeros, i);
        snprintf(ofile, 100, "./results/cuda-simple/distances/%s_%s%d.txt", size, zeros, i);

        // Create the output folder
        char *command = (char *) malloc(100 * sizeof(char));
        snprintf(command, 100, "mkdir -p ./results/cuda-simple/distances");
        system(command);

        if (VERBOSE) {
            printf("Output file: %s\n", ofile);
            printf("Graph to read file: %s\n", gfile);
        }

        // Read graph
        Graph1D graph = read_graph1D(gfile);

        
        // Start the timer
        start = clock();

        int n = graph.V; // Vertices
        int src = 0;

        if (n > num_threads) {tile_size = n / num_threads + 1;} 

        int *dist     = (int *) malloc(n * sizeof(int));
        int *negative = (int *) malloc(sizeof(int));
        *negative = 0;
        
        // Initialize the distances array
        for (int j = 0; j < n; j++) {
            dist[j] = INF;
        }
        dist[src] = 0;
        // Source vertex
        dist[src] = 0;

        
        // Move pointers to cuda device
        int *d_adj, *d_dist, *d_negative;
        
        hipMalloc((void **)&d_adj, n * n * sizeof(int));
        hipMalloc((void **)&d_dist, n * sizeof(int));
        hipMalloc((void **)&d_negative, sizeof(int));

        hipMemcpy(d_adj, graph.adj, n * n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, dist, n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_negative, negative, sizeof(int), hipMemcpyHostToDevice);

    
        bellman_ford_tiled<<<1, num_threads>>>
                                    (d_adj, 
                                     d_dist, 
                                     n, 
                                     d_negative, 
                                     tile_size
                                    );
                                    
        // Copy back the results                        
        hipMemcpy(dist, d_dist, n * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(negative, d_negative, sizeof(int), hipMemcpyDeviceToHost);
        
        // Save results
        end = clock();

        // Progress bar
        if (!VERBOSE) { printf("\rIn progress %f%%", (float)(i+1)/atoi(n_graphs)*100);
        fflush(stdout);}
       
        double elapsed_time = double(end - start) / CLOCKS_PER_SEC;
        avg_time += elapsed_time;

        if(VERBOSE) printf("Elapsed time: %f ", elapsed_time);
 
        if (*negative) {
            save_negative(ofile);
        }
        else {
            save_dist_array(dist, n, ofile);
        }
    
        FILE *fp;
        fp = fopen("./results/cuda-simple/times.txt", "a");
        fprintf(fp, "%s %.5f %d\n", size, elapsed_time, num_threads);
        fclose(fp);
        
        // Free memory
        free(dist); free(negative); free(graph.adj);
        hipFree(d_dist); hipFree(d_negative); hipFree(d_adj);
        
        // Check for errors
        hipError_t error;
        error = hipGetLastError();
        const char *error_str = hipGetErrorString(error);
        printf("%s\n", error_str);
    }

    printf("\n");
    printf("Average time: %f\n", avg_time / atoi(n_graphs));

    free(gfile); free(ofile); free(zeros);


    printf("Done\n");

    
    return 0;

}